/*
Hello world of wave propagation in CUDA. FDTD acoustic wave propagation in homogeneous medium. Second order in space and time 
*/

#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include "string.h"
/*
Add this to c_cpp_properties.json if linting isn't working for cuda libraries
"includePath": [
                "/usr/local/cuda-9.0/targets/x86_64-linux/include",
                "${workspaceFolder}/**"
            ],
*/          
#include "hip/hip_runtime.h"



// Check error codes for CUDA functions
#define CHECK(call)                                                            \
{                                                                              \
    hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

#define PI      3.14159265359
#define PAD     4
#define PAD2    8
#define a0     -3.0124472f
#define a1      1.7383092f
#define a2     -0.2796695f
#define a3      0.0547837f
#define a4     -0.0073118f

#define BDIMX  32
#define BDIMY  32


// Allocate the constant device memory
__constant__ float c_coef[5];       /* coefficients for 8th order fd */
__constant__ int c_isrc;            /* source location, ox */
__constant__ int c_jsrc;            /* source location, oz */
__constant__ int c_nx;              /* x dim */
__constant__ int c_ny;              /* y dim */
__constant__ int c_nt;              /* time steps */

// Add source wavelet
__global__ void kernel_add_wavelet(float *d_u1, float *d_wavelet, int it)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * c_nx + ix;

    if ((ix == c_isrc) && (iy == c_jsrc)) 
    {
        d_u1[idx] += d_wavelet[it];
        printf("%d %f\n",it, d_wavelet[it]);
    }
}

// FD kernel
__global__ void kernel_2dfd(float *d_u1, float *d_u2, float *d_vp)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * c_nx + ix;


    // Load blockinto memory
    __shared__ float patch[BDIMX + PAD2][BDIMX + PAD2];
    // printf("%d\n",idx);
    
}

int main( int argc, char *argv[])
{
    // Print out name of the main GPU
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, 0));
    printf("%s\t%d.%d:\n", deviceProp.name, deviceProp.major, deviceProp.minor);
    printf("%lu GB:\t total Global memory (gmem)\n", deviceProp.totalGlobalMem/1024/1024/1000);
    printf("%lu MB:\t total Constant memory (cmem)\n", deviceProp.totalConstMem/1024);
    printf("%lu MB:\t total Shared memory per block (smem)\n", deviceProp.sharedMemPerBlock/1024);
    printf("%d:\t total threads per block\n", deviceProp.maxThreadsPerBlock);
    printf("%d:\t total registers per block\n", deviceProp.regsPerBlock);
    printf("%d:\t warp size\n", deviceProp.warpSize);
    printf("%d x %d x %d:\t max dims of block\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("%d x %d x %d:\t max dims of grid\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    CHECK(hipSetDevice(0));

    // Model dimensions
    int nx    = 1024;                       /* x dim */
    int ny    = 1024;                       /* z dim */

    // Add padding for derivatives
    nx += 2 * PAD;
    ny += 2 * PAD;

    size_t nxy = nx * ny;   
    size_t nbytes = nxy * sizeof(float);    /* bytes to store nx * ny */
    
    float dx = 1;                           /* m */
    float dy = dx;
    
    // Allocate memory for velocity model
    float _vp = 3300;                       /* m/s, p-wave velocity */
    float *h_vp;
    h_vp = (float *)malloc(nbytes);
    memset(h_vp, _vp, nbytes);              /* initiate h_vp with _vp */

    // Time stepping
    float t_total = 0.05;                   /* sec, total time of wave propagation */
    float dt = 0.7 * fmin(dx, dy) / _vp;    /* sec, time step assuming constant vp */
    int nt = round(t_total / dt);         /* number of time steps */

    // Source
    float f0 = 100.0;                        /* Hz, source dominant frequency */
    float t0 = 1.2 / f0;                    /* source padding to move wavelet from left of zero */

    float *h_wavelet, *h_time;
    h_time = (float *) malloc(nt * sizeof(float));
    h_wavelet = (float *) malloc(nt * sizeof(float));

    // Fill source waveform vecror
    float a = PI * PI * f0 * f0;            /* const for wavelet */
    for(size_t it = 0; it < nt; it++)
    {
        h_time[it] = it * dt;
        h_wavelet[it] = 1e10 * (1.0 - 2.0*a*pow(h_time[it] - t0, 2))*exp(-a*pow(h_time[it] - t0, 2));
        h_wavelet[it] *= dt * dt / (dx * dy);
    }

    // Allocate memory on device
    float *d_u1, *d_u2, *d_vp, *d_wavelet;
    CHECK(hipMalloc((void **) &d_u1, nbytes))          /* wavefield at t-1 */
    CHECK(hipMalloc((void **) &d_u2, nbytes))          /* wavefield at t-2 */
    CHECK(hipMalloc((void **) &d_vp, nbytes))          /* velocity model */
    CHECK(hipMalloc((void **) &d_wavelet, nbytes));    /* source term for each time step */
    
    CHECK(hipMemset(d_u1, 0, nbytes))
    CHECK(hipMemset(d_u2, 0, nbytes))
    CHECK(hipMemcpy(d_vp, h_vp, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_wavelet, h_wavelet, nbytes, hipMemcpyHostToDevice));

    float coef[] = {a0, a1, a2, a3, a4};
    int isrc = round((float) nx / 2);                 /* source location, ox */
    int jsrc = round((float) ny / 2);                 /* source location, oz */

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_coef), coef, 5 * sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_isrc), &isrc, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_jsrc), &jsrc, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nx), &nx, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_ny), &ny, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nt), &nt, sizeof(int)));


    // Setup kernel run
    dim3 block(BDIMX, BDIMY);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    
    printf("%i\tnt\n",nt);
    for(int it = 0; it < nt; it++)
    {
        kernel_add_wavelet<<<grid,block>>>(d_u1, d_wavelet, it);
        kernel_2dfd<<<grid,block>>>(d_u1, d_u2, d_vp);
    }
    
    free(h_vp);
    free(h_time);
    free(h_wavelet);

    CHECK(hipFree(d_u1));
    CHECK(hipFree(d_u2));
    CHECK(hipFree(d_vp));
    CHECK(hipFree(d_wavelet));

    CHECK(hipDeviceReset());



    return 0;
}
